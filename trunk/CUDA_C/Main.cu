#include "Constants.h"
#include "SEBS_kb_1.cuh"
#include "SEBS_EnergyBalance.cuh"
#include "SEBS_Daily_Evapotranspiration.cuh"
#include "AUXFunc.cuh"
#include <stdio.h>

/* NOTE: Skip to "Main Calculation Loop" to see how to use the SEBS functions. 
         Most of the code here is just to show how to read data and feed it to
         the calculation part. Depending on your program this part can change.
*/

int main(void)
{
	/* Defining Variables *****************************************************/
	/* Input Variables */
	int NData=dimx*dimy;
	unsigned long int memsize=NData*sizeof(float);
	unsigned long int memsize_int=NData*sizeof(int);
	float *fc, *LAI, *NDVI, *LST_K, *hc, *Zref, *Uref, *Pref;
	float *P0, *Ps, *Tref_K, *qa_ref, *Albedo, *Emissivity;
	float *SWd, *LWd, *hpbl, *SWd24;
	float *lat_rad, *Ta_av_K, *Ns;
	int *ComptMask;
	float day_angle=194.0;
	
	/* Output Variables */
	float *d0, *z0h, *z0m;
	float *Rn, *G0, *H, *LE, *EF, *re_i, *ustar, *H_DL, *H_WL;
	float *Rndaily, *Edaily;

	/* Input Variables on Device */
	float *fc_d, *LAI_d, *NDVI_d, *LST_K_d, *hc_d, *Zref_d, *Uref_d, *Pref_d;
	float *P0_d, *Ps_d, *Tref_K_d, *qa_ref_d, *Albedo_d, *Emissivity_d;
	float *SWd_d, *LWd_d, *hpbl_d, *SWd24_d;
	float *lat_rad_d, *Ta_av_K_d, *Ns_d;
	int *ComptMask_d;
	float *day_angle_d;
	
	/* Output Variables on Device*/
	float *d0_d, *z0h_d, *z0m_d;
	float *Rn_d, *G0_d, *H_d, *LE_d, *EF_d, *re_i_d, *ustar_d, *H_DL_d, *H_WL_d;
	float *Rndaily_d, *Edaily_d;

	/* General Variables */
	dim3 dimGrid((int)NData/nThreadsPerBlock+1); 
    dim3 dimBlock(nThreadsPerBlock);
    
    int DeviceCount;
    hipDeviceProp_t DeviceProp;

	/* Timing Variables */
	hipEvent_t start_event, stop_event;
	float ElapTime;
	/* End of Defining Variables *********************************************/

	/* Getting Some info regarding the GPU Device ****************************/
	hipGetDeviceCount(&DeviceCount);
	if (DeviceCount==0)
	{
		printf("FATAL ERROR: There is no CUDA-Enabled Device available!!!\n");
		return(-1);
	}
	else
	{
			printf("Number of GPU device available: %d\n",DeviceCount);
	}
	if (SelectedDevice>DeviceCount-1)
	{
		printf("FATAL ERROR: The selected device does not exist.\n");
		return(-1);
	}
	hipSetDevice(SelectedDevice);
	printf("Selected Device:\n");
	hipGetDeviceProperties(&DeviceProp,SelectedDevice);
	printf("Device Name: %s\n",DeviceProp.name);
	printf("Compute Capability: %d.%d\n",DeviceProp.major,DeviceProp.minor);
	printf("Number of MultiProcessors: %d\n",DeviceProp.multiProcessorCount);
	/* END OF Getting Some info regarding the GPU Device *********************/

	/* Allocating Memories ***************************************************/
	printf("Allocating Memory on host ... \n");
	CheckMemAllocationError(fc=(float *) malloc(memsize),"fc");
	CheckMemAllocationError(LAI=(float *) malloc(memsize),"LAI");
	CheckMemAllocationError(NDVI=(float *) malloc(memsize),"NDVI");
	CheckMemAllocationError(LST_K=(float *) malloc(memsize),"LST_K");
	CheckMemAllocationError(hc=(float *) malloc(memsize),"hc");
	CheckMemAllocationError(Zref=(float *) malloc(memsize),"Zref");
	CheckMemAllocationError(Uref=(float *) malloc(memsize),"Uref");
	CheckMemAllocationError(Pref=(float *) malloc(memsize),"Pref");
	CheckMemAllocationError(P0=(float *) malloc(memsize),"P0");
	CheckMemAllocationError(Ps=(float *) malloc(memsize),"Ps");
	CheckMemAllocationError(Tref_K=(float *) malloc(memsize),"Tref_K");
	CheckMemAllocationError(qa_ref=(float *) malloc(memsize),"qa_ref");
	CheckMemAllocationError(Albedo=(float *) malloc(memsize),"Albedo");
	CheckMemAllocationError(Emissivity=(float *) malloc(memsize),"Emissivity");
	CheckMemAllocationError(SWd=(float *) malloc(memsize),"SWd");
	CheckMemAllocationError(LWd=(float *) malloc(memsize),"LWd");
	CheckMemAllocationError(hpbl=(float *) malloc(memsize),"hpbl");
	CheckMemAllocationError(SWd24=(float *) malloc(memsize),"SWd24");
	CheckMemAllocationError(lat_rad=(float *) malloc(memsize),"lat_rad");
	CheckMemAllocationError(Ta_av_K=(float *) malloc(memsize),"Ta_av_K");
	CheckMemAllocationError(Ns=(float *) malloc(memsize),"Ns");
	CheckMemAllocationError(ComptMask=(int *) malloc(memsize_int),"ComptMask");

	CheckMemAllocationError(d0=(float *) malloc(memsize),"d0");
	CheckMemAllocationError(z0h=(float *) malloc(memsize),"z0h");
	CheckMemAllocationError(z0m=(float *) malloc(memsize),"z0m");
	CheckMemAllocationError(Rn=(float *) malloc(memsize),"Rn");
	CheckMemAllocationError(G0=(float *) malloc(memsize),"G0");
	CheckMemAllocationError(H=(float *) malloc(memsize),"H");
	CheckMemAllocationError(LE=(float *) malloc(memsize),"LE");
	CheckMemAllocationError(EF=(float *) malloc(memsize),"EF");
	CheckMemAllocationError(re_i=(float *) malloc(memsize),"re_i");
	CheckMemAllocationError(ustar=(float *) malloc(memsize),"ustar");
	CheckMemAllocationError(H_DL=(float *) malloc(memsize),"H_DL");
	CheckMemAllocationError(H_WL=(float *) malloc(memsize),"H_WL");
	CheckMemAllocationError(Rndaily=(float *) malloc(memsize),"Rndaily");
	CheckMemAllocationError(Edaily=(float *) malloc(memsize),"Edaily");
	printf("Done.\n");

	printf("Allocating Memory on device ... \n");
	hipMalloc( (void **) &fc_d, memsize);
	checkCUDAError("allocating memory for fc_d");
	hipMalloc( (void **) &LAI_d, memsize);
	checkCUDAError("allocating memory for LAI_d");
	hipMalloc( (void **) &NDVI_d, memsize);
	checkCUDAError("allocating memory for NDVI_d");
	hipMalloc( (void **) &LST_K_d, memsize);
	checkCUDAError("allocating memory for LST_K_d");
	hipMalloc( (void **) &hc_d, memsize);
	checkCUDAError("allocating memory for hc_d");
	hipMalloc( (void **) &Zref_d, memsize);
	checkCUDAError("allocating memory for Zref_d");
	hipMalloc( (void **) &Uref_d, memsize);
	checkCUDAError("allocating memory for Uref_d");
	hipMalloc( (void **) &Pref_d, memsize);
	checkCUDAError("allocating memory for Pref_d");
	hipMalloc( (void **) &P0_d, memsize);
	checkCUDAError("allocating memory for P0_d");
	hipMalloc( (void **) &Ps_d, memsize);
	checkCUDAError("allocating memory for Ps_d");
	hipMalloc( (void **) &Tref_K_d, memsize);
	checkCUDAError("allocating memory for Tref_K_d");
	hipMalloc( (void **) &qa_ref_d, memsize);
	checkCUDAError("allocating memory for qa_ref_d");
	hipMalloc( (void **) &Albedo_d, memsize);
	checkCUDAError("allocating memory for Albedo_d");
	hipMalloc( (void **) &Emissivity_d, memsize);
	checkCUDAError("allocating memory for Emissivity_d");
	hipMalloc( (void **) &SWd_d, memsize);
	checkCUDAError("allocating memory for SWd_d");
	hipMalloc( (void **) &LWd_d, memsize);
	checkCUDAError("allocating memory for LWd_d");
	hipMalloc( (void **) &hpbl_d, memsize);
	checkCUDAError("allocating memory for hpbl_d");
	hipMalloc( (void **) &SWd24_d, memsize);
	checkCUDAError("allocating memory for SWd24_d");
	hipMalloc( (void **) &lat_rad_d, memsize);
	checkCUDAError("allocating memory for lat_rad_d");
	hipMalloc( (void **) &Ta_av_K_d, memsize);
	checkCUDAError("allocating memory for Ta_av_K_d");
	hipMalloc( (void **) &Ns_d, memsize);
	checkCUDAError("allocating memory for Ns_d");
	hipMalloc( (void **) &day_angle_d, sizeof(float));
	checkCUDAError("allocating memory for day_angle_d");
	hipMalloc( (void **) &ComptMask_d, memsize_int);
	checkCUDAError("allocating memory for ComptMask_d");

	hipMalloc( (void **) &d0_d, memsize);
	checkCUDAError("allocating memory for d0_d");
	hipMalloc( (void **) &z0h_d, memsize);
	checkCUDAError("allocating memory for z0h_d");
	hipMalloc( (void **) &z0m_d, memsize);
	checkCUDAError("allocating memory for z0m_d");
	hipMalloc( (void **) &Rn_d, memsize);
	checkCUDAError("allocating memory for Rn_d");
	hipMalloc( (void **) &G0_d, memsize);
	checkCUDAError("allocating memory for G0_d");
	hipMalloc( (void **) &H_d, memsize);
	checkCUDAError("allocating memory for H_d");
	hipMalloc( (void **) &LE_d, memsize);
	checkCUDAError("allocating memory for LE_d");
	hipMalloc( (void **) &EF_d, memsize);
	checkCUDAError("allocating memory for EF_d");
	hipMalloc( (void **) &re_i_d, memsize);
	checkCUDAError("allocating memory for re_i_d");
	hipMalloc( (void **) &ustar_d, memsize);
	checkCUDAError("allocating memory for ustar_d");
	hipMalloc( (void **) &H_DL_d, memsize);
	checkCUDAError("allocating memory for H_DL_d");
	hipMalloc( (void **) &H_WL_d, memsize);
	checkCUDAError("allocating memory for H_WL_d");
	hipMalloc( (void **) &Rndaily_d, memsize);
	checkCUDAError("allocating memory for Rndaily_d");
	hipMalloc( (void **) &Edaily_d, memsize);
	checkCUDAError("allocating memory for Edaily_d");	
	printf("Done.\n");
	/* END of Allocating Memories ********************************************/
	
	/* Creating CUDA event to time the code **********************************/
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
	/* END OF Creating CUDA event to time the code ***************************/

	/* Loading the input Variables *******************************************/
	ReadNetCDF(fc, "fc", "../InputData/sfc.nc");
	ReadNetCDF(LAI, "LAI", "../InputData/sLAI.nc");
	ReadNetCDF(NDVI, "NDVI", "../InputData/sNDVI.nc");
	ReadNetCDF(LST_K, "Ts", "../InputData/sTs.nc");
	ReadNetCDF(hc, "hc", "../InputData/shc.nc");
	ReadNetCDF(Zref, "Zref", "../InputData/Zref.nc");
	ReadNetCDF(Uref, "Uref", "../InputData/sUref.nc");
	ReadNetCDF(Pref, "Pressure", "../InputData/sPressure.nc");
	ReadNetCDF(P0, "prmsl", "../InputData/sprmsl.nc");
	ReadNetCDF(Ps, "Pressure", "../InputData/sPressure.nc");
	ReadNetCDF(Tref_K, "Ta_ref", "../InputData/sTa_ref.nc");
	ReadNetCDF(qa_ref, "qa_ref", "../InputData/sqa_ref.nc");
	ReadNetCDF(Albedo, "Albedo", "../InputData/sAlbedo.nc");
	ReadNetCDF(Emissivity, "Emissivity", "../InputData/sEmissivity.nc");
	ReadNetCDF(SWd, "SWd", "../InputData/sSWd.nc");
	ReadNetCDF(LWd, "LWd", "../InputData/sLWd.nc");
	ReadNetCDF(hpbl, "hpbl", "../InputData/shpbl.nc");
	ReadNetCDF(SWd24, "SWd24", "../InputData/sSWd24.nc");
	ReadNetCDF(lat_rad, "lat", "../InputData/slat.nc");
	ReadNetCDF(Ta_av_K, "Ta_avg", "../InputData/sTa_avg.nc");
	ReadNetCDF(Ns, "Ns", "../InputData/sNs.nc");
	ReadNetCDF_int(ComptMask, "ComptMask", "../InputData/sComptMask.nc");
	/* END OF Loading the input Variables ************************************/	

	/* Starting the Timer ****************************************************/
	printf("Starting the Timer ...");
	hipEventRecord( start_event, 0 );
	checkCUDAError("Starting the Timer.");
	printf(" done.\n");
	
	/* Uploading Data to Device **********************************************/
	printf("Uploading Data to Device Memory ...\n");
	hipMemcpy( fc_d, fc, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading fc");
	hipMemcpy( LAI_d, LAI, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading LAI");
	hipMemcpy( NDVI_d, NDVI, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading NDVI");
	hipMemcpy( LST_K_d, LST_K, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading LST_K");
	hipMemcpy( hc_d, hc, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading hc");
	hipMemcpy( Zref_d, Zref, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Zref");
	hipMemcpy( Uref_d, Uref, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Uref");
	hipMemcpy( Pref_d, Pref, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Pref");
	hipMemcpy( P0_d, P0, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading P0");
	hipMemcpy( Ps_d, Ps, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Ps");
	hipMemcpy( Tref_K_d, Tref_K, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Tref_K");
	hipMemcpy( qa_ref_d, qa_ref, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading qa_ref");
	hipMemcpy( Albedo_d, Albedo, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Albedo");
	hipMemcpy( Emissivity_d, Emissivity, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Emissivity");
	hipMemcpy( SWd_d, SWd, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading SWd");
	hipMemcpy( LWd_d, LWd, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading LWd");
	hipMemcpy( hpbl_d, hpbl, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading hpbl");
	hipMemcpy( SWd24_d, SWd24, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading SWd24");
	hipMemcpy( lat_rad_d, lat_rad, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading lat_rad");
	hipMemcpy( Ta_av_K_d, Ta_av_K, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Ta_av_K");
	hipMemcpy( Ns_d, Ns, memsize, hipMemcpyHostToDevice );
	checkCUDAError("Uploading Ns");
	hipMemcpy( ComptMask_d, ComptMask, memsize_int, hipMemcpyHostToDevice );
	checkCUDAError("Uploading ComptMask");
	hipMemcpy( day_angle_d, &day_angle, sizeof(float), hipMemcpyHostToDevice );
	checkCUDAError("Uploading day_angle");
	printf(" done.\n");
	/* END OF Uploading Data to Device ***************************************/
	
/*********************************************************************************************/
/*********************************** Main Calculation Loop ***********************************/
/*********************************************************************************************/

	printf("-- running SEBS_kb_1 \n");
	SEBS_kb_1_Kernel<<< dimGrid, dimBlock >>>(ComptMask_d,fc_d,LAI_d,NDVI_d,
											  LST_K_d,hc_d,Zref_d,Uref_d,
											  Pref_d,P0_d,Ps_d,Tref_K_d,
											  qa_ref_d,z0m_d,d0_d,z0h_d,NData);
	checkCUDAError("Running SEBS_kb_1_Kernel");
	hipDeviceSynchronize();
		
	printf("-- SEBS_EnergyBalance ...\n");
	SEBS_EnergyBalance_Kernel<<< dimGrid, dimBlock >>>(ComptMask_d,d0_d,z0m_d,z0h_d,
									fc_d,LAI_d,hc_d,Albedo_d,Emissivity_d,LST_K_d,NDVI_d,
									SWd_d,LWd_d,hpbl_d,Zref_d,Tref_K_d,Uref_d,qa_ref_d,
									Pref_d,Ps_d,P0_d,
									Rn_d,G0_d,H_d,LE_d,EF_d,re_i_d,ustar_d,H_DL_d,H_WL_d,NData);		
	checkCUDAError("Running SEBS_EnergyBalance_Kernel");
	hipDeviceSynchronize();
	
	printf("-- SEBS_Daily_Evapotranspiration\n");
	SEBS_Daily_Evapotranspiration_Kernel<<< dimGrid, dimBlock >>>(
							ComptMask_d,day_angle_d,lat_rad_d,
							Albedo_d,SWd24_d,Ta_av_K_d,EF_d, Ns_d,
							Rndaily_d,Edaily_d,NData); 
	checkCUDAError("Running SEBS_Daily_Evapotranspiration_Kernel");
	hipDeviceSynchronize();
	printf("-- Calculationg is done.\n");
/*********************************************************************************************/
/******************************** END OF Main Calculation Loop *******************************/
/*********************************************************************************************/

	/* Downloading Data to Device ********************************************/
	printf("-- Transfering data from Device to Host ...\n");
	hipMemcpy( d0, d0_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading d0");
	hipMemcpy( z0m, z0m_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading z0m");
	hipMemcpy( z0h, z0h_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading z0h");
	hipMemcpy( Rn, Rn_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading Rn");
	hipMemcpy( G0, G0_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading G0");
	hipMemcpy( H, H_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading H");
	hipMemcpy( LE, LE_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading LE");
	hipMemcpy( EF, EF_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading EF");
	hipMemcpy( re_i, re_i_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading re_i");
	hipMemcpy( ustar, ustar_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading ustar");
	hipMemcpy( H_DL, H_DL_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading H_DL");
	hipMemcpy( H_WL, H_WL_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading H_WL");
	hipMemcpy( Rndaily, Rndaily_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading Rndaily");
	hipMemcpy( Edaily, Edaily_d, memsize, hipMemcpyDeviceToHost );
	checkCUDAError("downloading Edaily");
	/* END OF Downloading Data to Device *************************************/

	/* Stoping the Timer *****************************************************/
	hipEventRecord( stop_event, 0 );
	hipEventSynchronize( stop_event );
	hipEventElapsedTime( &ElapTime, start_event, stop_event );
	printf("Total Calculation Time including memory transfer [ms]: %f\n",ElapTime);

	/* Writting the Output ***************************************************/
	WriteOutput(z0m,"./Output/z0m.dat",NData);
	WriteOutput(z0h,"./Output/z0h.dat",NData);
	WriteOutput(d0,"./Output/d0.dat",NData);
	WriteOutput(Rn,"./Output/Rn.dat",NData);
	WriteOutput(G0,"./Output/G0.dat",NData);
	WriteOutput(H,"./Output/H.dat",NData);
	WriteOutput(LE,"./Output/LE.dat",NData);
	WriteOutput(EF,"./Output/EF.dat",NData);
	WriteOutput(re_i,"./Output/re_i.dat",NData);
	WriteOutput(ustar,"./Output/ustar.dat",NData);
	WriteOutput(H_DL,"./Output/H_DL.dat",NData);
	WriteOutput(H_WL,"./Output/H_WL.dat",NData);
	WriteOutput(Rndaily,"./Output/Rndaily.dat",NData);
	WriteOutput(Edaily,"./Output/Edaily.dat",NData);
	/* END OF Writting the Output ********************************************/

	/* Freeing Memory ********************************************************/	
	hipFree(ComptMask);
	hipFree(fc_d);
	hipFree(LAI_d);
	hipFree(NDVI_d);
	hipFree(LST_K_d);
	hipFree(hc_d);
	hipFree(Zref_d);
	hipFree(Uref_d);
	hipFree(Pref_d);
	hipFree(P0_d);
	hipFree(Ps_d);
	hipFree(Tref_K_d);
	hipFree(qa_ref_d);
	hipFree(Albedo_d);
	hipFree(Emissivity_d);
	hipFree(SWd_d);
	hipFree(LWd_d);
	hipFree(hpbl_d);
	hipFree(day_angle_d);
	hipFree(lat_rad_d);
	hipFree(SWd24_d);
	hipFree(Ta_av_K_d);
	hipFree(Ns_d); 

	hipFree(d0_d);
	hipFree(z0h_d);
	hipFree(z0m_d);
	hipFree(Rn_d);
	hipFree(G0_d);
	hipFree(H_d);
	hipFree(LE_d);
	hipFree(EF_d);
	hipFree(re_i_d);
	hipFree(ustar_d);
	hipFree(H_DL_d);
	hipFree(H_WL_d);
	hipFree(Rndaily_d);
	hipFree(Edaily_d);
	/* END OF Freeing Memory ********************************************************/	

	hipEventDestroy(start_event);
	hipEventDestroy(stop_event);

	printf("All done!\n");
	return(0);
}


